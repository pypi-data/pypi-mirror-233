#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_Identities32_to_Identities64(T* toptr,
                                     const C* fromptr,
                                     int64_t length,
                                     int64_t width,
                                     uint64_t invocation_index,
                                     uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < (length * width)) {
      toptr[thread_id] = (int64_t)(fromptr[thread_id]);
    }
  }
}
