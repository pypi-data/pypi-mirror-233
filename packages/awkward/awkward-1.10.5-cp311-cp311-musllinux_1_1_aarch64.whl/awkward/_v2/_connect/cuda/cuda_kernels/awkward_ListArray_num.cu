#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C, typename U>
__global__ void
awkward_ListArray_num(T* tonum,
                      const C* fromstarts,
                      const U* fromstops,
                      int64_t length,
                      uint64_t invocation_index,
                      uint64_t* err_code) {
  int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (err_code[0] == NO_ERROR) {
    if (thread_id < length) {
      int64_t start = fromstarts[thread_id];
      int64_t stop = fromstops[thread_id];
      tonum[thread_id] = (C)(stop - start);
    }
  }
}
